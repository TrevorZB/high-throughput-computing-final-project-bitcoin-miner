#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <chrono>
#include "stencil.cuh"


void randomize_array(float *a, int size, float start, float stop)
{
    // randomize the seed, create distribution
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937 gen(seed);
    std::uniform_real_distribution<float> dist(start, stop);

    for (int i = 0; i < size; i++)
    {
        a[i] = dist(gen);
    }
}

int main(int argc, char *argv[])
{
    int n = atoi(argv[1]);
    int R = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);

    float *image_host = new float[n];
    float *image_dev;
    float *mask_host = new float[n];
    float *mask_dev;
    float *output_host = new float[n];
    float *output_dev;

    // randomize array image, pass to device
    hipMalloc((void **)&image_dev, sizeof(float) * n);
    randomize_array(image_host, n, -1.0, 1.0);
    hipMemcpy(image_dev, image_host, sizeof(float) * n, hipMemcpyHostToDevice);

    // randomize array mask, pass to device
    hipMalloc((void **)&mask_dev, sizeof(float) * n);
    randomize_array(mask_host, n, -1.0, 1.0);
    hipMemcpy(mask_dev, mask_host, sizeof(float) * n, hipMemcpyHostToDevice);

    // create output array on the device
    hipMalloc((void **)&output_dev, sizeof(float) * n);

    // timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    stencil(image_dev, mask_dev, output_dev, n, R, threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // calc time in ms
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy output_dev to output_host
    hipMemcpy(output_host, output_dev, sizeof(float) * n, hipMemcpyDeviceToHost);

    printf("%f\n", output_host[n - 1]);
    printf("%f\n", ms);

    // free all memory on host and device
    delete[] image_host;
    delete[] mask_host;
    delete[] output_host;
    hipFree(image_dev);
    hipFree(mask_dev);
    hipFree(output_dev);
}