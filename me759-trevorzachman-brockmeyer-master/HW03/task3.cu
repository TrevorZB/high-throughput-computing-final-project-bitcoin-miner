#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <chrono>
#include "vscale.cuh"

#include <iostream>

void randomize_array(float *a, int size, float start, float stop)
{
    // randomize the seed, create distribution
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937 gen(seed);
    std::uniform_real_distribution<float> dist(start, stop);

    for (int i = 0; i < size; i++)
    {
        a[i] = dist(gen);
    }
}

int main(int argc, char *argv[])
{
    int n = atoi(argv[1]);
    int threads_per_block = 512;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    float *a_host = new float[n];
    float *a_dev;
    float *b_host = new float[n];
    float *b_dev;

    // randomize array a, pass to device
    hipMalloc((void **)&a_dev, sizeof(float) * n);
    randomize_array(a_host, n, -10.0, 10.0);
    hipMemcpy(a_dev, a_host, sizeof(float) * n, hipMemcpyHostToDevice);
    
    // randomize array b, pass to device
    hipMalloc((void **)&b_dev, sizeof(float) * n);
    randomize_array(b_host, n, 0.0, 1.0);
    hipMemcpy(b_dev, b_host, sizeof(float) * n, hipMemcpyHostToDevice);

    // timing variables
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vscale<<<blocks_per_grid, threads_per_block>>>(a_dev, b_dev, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // calc time in ms
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(b_host, b_dev, sizeof(float) * n, hipMemcpyDeviceToHost);

    printf("%f\n", ms);
    printf("%f\n", b_host[0]);
    printf("%f\n", b_host[n - 1]);

    delete[] a_host;
    delete[] b_host;
    hipFree(a_dev);
    hipFree(b_dev);
}